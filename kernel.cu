#include "hip/hip_runtime.h"
texture<unsigned int, 2, hipReadModeElementType> tex;

__global__ void interpolate(unsigned int * __restrict__ d_result, const int M, const int N, const float sigma_d, const float sigma_r)
{
    const int i = threadIdx.x + blockDim.x * blockIdx.x;
    const int j = threadIdx.y + blockDim.y * blockIdx.y;


    if ((i<M)&&(j<N)) {
        float s = 0;
        float c = 0;
        for (int l = i-1; l <= i+1; l++){
            for (int k = j-1; k <= j+1; k++){
                float img1 = tex2D(tex, k, l)/255;
                float img2 = tex2D(tex, i, j)/255;
                float g = exp(-(pow(k - i, 2) + pow(l - j, 2)) / pow(sigma_d, 2));
                float r = exp(-pow((img1 - img2)*255, 2) / pow(sigma_r, 2));
                c += g*r;
                s += g*r*tex2D(tex, k, l);
            }
        }
        d_result[i*N + j] = s / c;
    }


}
